#include "hip/hip_runtime.h"
#include "gpu_descracker.cuh"
#include <cstring>
#include <cstdio>

#define BLOCKSIZE 60

__device__ char gpu_alphabet[] = { 0, 'a', 'b', 'c'};

__device__ void gpu_word_for(int num, block_t* word, int alphabets)
{
	int sub = 1;
	int exp = 1;

	alphabets--;
	((char*)word)[7] = gpu_alphabet[((num - sub) >= 0)*(((num - sub) / exp) % alphabets + 1)];
	exp *= alphabets;
	sub += exp;
	((char*)word)[6] = gpu_alphabet[((num - sub) >= 0)*(((num - sub) / exp) % alphabets + 1)];
	exp *= alphabets;
	sub += exp;
	((char*)word)[5] = gpu_alphabet[((num - sub) >= 0)*(((num - sub) / exp) % alphabets + 1)];
	exp *= alphabets;
	sub += exp;
	((char*)word)[4] = gpu_alphabet[((num - sub) >= 0)*(((num - sub) / exp) % alphabets + 1)];
	exp *= alphabets;
	sub += exp;
	((char*)word)[3] = gpu_alphabet[((num - sub) >= 0)*(((num - sub) / exp) % alphabets + 1)];
	exp *= alphabets;
	sub += exp;
	((char*)word)[2] = gpu_alphabet[((num - sub) >= 0)*(((num - sub) / exp) % alphabets + 1)];
	exp *= alphabets;
	sub += exp;
	((char*)word)[1] = gpu_alphabet[((num - sub) >= 0)*(((num - sub) / exp) % alphabets + 1)];
	exp *= alphabets;
	sub += exp;
	((char*)word)[0] = gpu_alphabet[((num - sub) >= 0)*(((num - sub) / exp) % alphabets + 1)];
}

__host__ __device__ void gpu_fast_text_to_block(const char* text, block_t* block)
{
	*block = (0x00ffffffffffffff & *block) | (((block_t)(text[0])) << 56);
	*block = (0xff00ffffffffffff & *block) | (((block_t)(text[1])) << 48);
	*block = (0xffff00ffffffffff & *block) | (((block_t)(text[2])) << 40);
	*block = (0xffffff00ffffffff & *block) | (((block_t)(text[3])) << 32);
	*block = (0xffffffff00ffffff & *block) | (((block_t)(text[4])) << 24);
	*block = (0xffffffffff00ffff & *block) | (((block_t)(text[5])) << 16);
	*block = (0xffffffffffff00ff & *block) | (((block_t)(text[6])) << 8);
	*block = (0xffffffffffffff00 & *block) | ((block_t)(text[7]));
}

__global__ void gpu_des_crack_kernel(block_t* msg, block_t* lastWord, int keyshift, int* valid, block_t* keys)
{
	int keynum = blockDim.x*blockIdx.x + threadIdx.x;
	int val = 0;
	block_t tmpmsg, tmpkey, wordnum, encmsg; 
	gpu_word_for(keynum, &keys[keynum], 4);
	/*tmpmsg = 0;
	wordnum = 0;
	while (tmpmsg != *lastWord)
	{
		gpu_word_for(wordnum++, &tmpmsg, 4);

		encmsg = tmpmsg;
		gpu_des_encrypt(&encmsg, 1, tmpkey);
		if (encmsg == *msg)
			val = 1;
	}*/
	valid[keynum] = val;
}

block_t gpu_des_crack(block_t msg)
{
    hipError_t cudaStatus;
	block_t lastWord;
	int alphabets = sizeof(gpu_alphabet);
	int *dev_valid;
	block_t *dev_keys, *dev_message, *dev_lastWord;
	char mess[9];
	block_t keys[BLOCKSIZE];
	int valid[BLOCKSIZE];
	
	cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return 0;
    }

	cudaStatus = hipMalloc((void**)&dev_keys, BLOCKSIZE * sizeof(block_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return 0;
    }

	cudaStatus = hipMalloc((void**)&dev_valid, BLOCKSIZE * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
		hipFree(dev_keys);
        return 0;
    }

	cudaStatus = hipMalloc((void**)&dev_lastWord, sizeof(block_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
		hipFree(dev_keys);
		hipFree(dev_valid);
        return 0;
    }

	cudaStatus = hipMalloc((void**)&dev_message, sizeof(block_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
		hipFree(dev_keys);
		hipFree(dev_valid);
		hipFree(dev_lastWord);
        return 0;
    }

	cudaStatus = hipMemcpy(dev_lastWord, &lastWord, sizeof(block_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(dev_message, &msg, sizeof(block_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	memset(mess, 0, 9);
	memset(mess, 'c', 8);
	gpu_fast_text_to_block(mess, &lastWord);

	gpu_des_crack_kernel<<<1, BLOCKSIZE>>>(dev_message, dev_lastWord, 0, dev_valid, dev_keys);

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

	cudaStatus = hipMemcpy(keys, dev_keys, BLOCKSIZE * sizeof(block_t), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy keys failed!");
        goto Error;
    }
	cudaStatus = hipMemcpy(valid, dev_valid, BLOCKSIZE * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy valid failed!");
        goto Error;
    }

	for(int i=0; i<BLOCKSIZE; i++)
		printf("%d, %016llx\n", valid[i], keys[i]);

Error:
	hipFree(dev_keys);
	hipFree(dev_valid);
	hipFree(dev_lastWord);
	hipFree(dev_message);
	return 0;
}