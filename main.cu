#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <stdint.h>
#include <time.h>
#include "descommon.h"
#include "descracker.h"
#include "gpu_descracker.cuh"
#include "hip/hip_runtime.h"
#include ""

#define MSG_LEN 1

int main(int argc, char** argv)
{
	block_t key;
	block_t msg[MSG_LEN];
	block_t cracked_key;
	int i;
	clock_t started, finished;
	int millis, sec, min, h;
	
	memset(msg, 0, sizeof(msg));
	text_to_block("bcc", msg);
	text_to_block("bcc", &key);

	printf("Key: %016lx\n", key);
	des_encrypt(msg, MSG_LEN, key);
	for (i = 0; i < MSG_LEN; i++)
		printf("%016lx ", msg[i]);
	printf("\n");

	started = clock();
	cracked_key = gpu_des_crack(msg[0]);
	finished = clock();
	printf("\nCracked key: %016lx \n", cracked_key);
	millis = ((finished - started) * 1000) / CLOCKS_PER_SEC;
	sec = millis / 1000;
	min = sec / 60;
	h = min / 60;
	printf("Calculations time: %02d:%02d:%02d:%03d\n", h, min%60, sec%60, millis%1000);

	system("pause");
	return 0;
}
